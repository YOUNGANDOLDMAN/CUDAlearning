#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "aux.h"

typedef float FLOAT;

/* host, add */
void vec_add_host(FLOAT *x, FLOAT *y, FLOAT *z, int N);

/* device function */
__global__ void vec_add(FLOAT *x, FLOAT *y, FLOAT *z, int N)//和c语言一样的封装函数
{
    /* 1D block */
    int idx = get_tid();//得到线程的全局编号
    if (idx < N) z[idx] = z[idx] + y[idx] + x[idx];//函数运算
}

void vec_add_host(FLOAT *x, FLOAT *y, FLOAT *z, int N)//有意使用c语言，对比纯c语言和使用cuda的时间对比
{
    int i;

    for (i = 0; i < N; i++) z[i] = z[i] + y[i] + x[i];
}

int main()
{
    int N = 20000000;//确定两千万个浮点数
    int nbytes = N * sizeof(FLOAT);//定义两千万个浮点数数组所需要的内存空间

    /* 1D block */
    int bs = 256;//一个block里面有256个线程，还是一维

    /* 2D grid */
    int s = ceil(sqrt((N + bs - 1.) / bs));
    dim3 grid = dim3(s, s);

    FLOAT *dx = NULL, *hx = NULL;
    FLOAT *dy = NULL, *hy = NULL;
    FLOAT *dz = NULL, *hz = NULL;

    int itr = 30;
    int i;
    double th, td;

    /* allocate GPU mem */
    hipMalloc((void **)&dx, nbytes);
    hipMalloc((void **)&dy, nbytes);
    hipMalloc((void **)&dz, nbytes);

    if (dx == NULL || dy == NULL || dz == NULL) {
        printf("couldn't allocate GPU memory\n");
        return -1;
    }

    printf("allocated %.2f MB on GPU\n", nbytes / (1024.f * 1024.f));

    /* alllocate CPU mem */
    hx = (FLOAT *) malloc(nbytes);
    hy = (FLOAT *) malloc(nbytes);
    hz = (FLOAT *) malloc(nbytes);

    if (hx == NULL || hy == NULL || hz == NULL) {
        printf("couldn't allocate CPU memory\n");
        return -2;
    }
    printf("allocated %.2f MB on CPU\n", nbytes / (1024.f * 1024.f));

    /* init */
    for (i = 0; i < N; i++) {
        hx[i] = 1;
        hy[i] = 1;
        hz[i] = 1;
    }

    /* copy data to GPU */
    hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(dy, hy, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(dz, hz, nbytes, hipMemcpyHostToDevice);

    /* call GPU */
    hipDeviceSynchronize();
    td = get_time();
    
    for (i = 0; i < itr; i++) vec_add<<<grid, bs>>>(dx, dy, dz, N);

    hipDeviceSynchronize();
    td = get_time() - td;

    /* CPU */
    th = get_time();
    for (i = 0; i < itr; i++) vec_add_host(hx, hy, hz, N);
    th = get_time() - th;

    printf("GPU time: %e, CPU time: %e, speedup: %g\n", td, th, th / td);

    hipFree(dx);
    hipFree(dy);
    hipFree(dz);

    free(hx);
    free(hy);
    free(hz);

    return 0;
}
